#include "hip/hip_runtime.h"
#include <algorithm>
#include <fstream>
#include <iostream>
#include <cmath>
#include <map>
#include <stdio.h>
#include <stdlib.h>
#include <string>

#include <cusp/io/matrix_market.h>
#include <cusp/csr_matrix.h>
#include <cusp/multiply.h>
#include <cusp/blas.h>
#include <cusp/detail/format_utils.h>
#include <cusp/detail/host/convert.h>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include <cusparse/common.h>
#include <cusparse/timer.h>
#include "hipsparse.h"

using std::endl;
using std::cerr;
using std::cout;

// -----------------------------------------------------------------------------
// -----------------------------------------------------------------------------
#ifdef WIN32
#   define ISNAN(A)  (_isnan(A))
#else
#   define ISNAN(A)  (isnan(A))
#endif

enum TestColor {COLOR_NO = 0,
                COLOR_RED,
                COLOR_GREEN} ;

class OutputItem
{
public:
	OutputItem(std::ostream &o): m_o(o), m_additional_item_count(19) {}

	int           m_additional_item_count;

	template <typename T>
	void operator() (T item, TestColor c = COLOR_NO) {
		m_o << "<td style=\"border-style: inset;\">\n";
		switch (c)
		{
			case COLOR_RED:
				m_o << "<p> <FONT COLOR=\"Red\">" << item << " </FONT> </p>\n";
				break;

			case COLOR_GREEN:
				m_o << "<p> <FONT COLOR=\"Green\">" << item << " </FONT> </p>\n";
				break;

			default:
				m_o << "<p> " << item << " </p>\n";
				break;
		}
		m_o << "</td>\n";
	}
private:
	std::ostream &m_o;
};

int main(int argc, char **argv)
{
	if (argc < 2) {
		cerr << "Usage: ./example MATRIX_MARKET_FILE_NAME" << endl;
		return 1;
	}

	cusp::csr_matrix<int, double, cusp::device_memory> Ad_cusp;
	cusp::io::read_matrix_market_file(Ad_cusp, argv[1]);

	cusparse::CuSparseCsrMatrixD A(Ad_cusp.row_offsets, Ad_cusp.column_indices, Ad_cusp.values);
	cusparse::CuSparseCsrMatrixD Abak(Ad_cusp.row_offsets, Ad_cusp.column_indices, Ad_cusp.values);

	thrust::device_vector<double> x(A.m_n, 1.0);
	thrust::device_vector<double> y;
	thrust::device_vector<double> x_new(A.m_n);

	// Name of matrix
	OutputItem outputItem(cout);

	cout << "<tr valign=top>" << endl;

	// Name of matrix
	{
		std::string fileMat = argv[1];
		int i;
		for (i = fileMat.size()-1; i>=0 && fileMat[i] != '/' && fileMat[i] != '\\'; i--);
		i++;
		fileMat = fileMat.substr(i);

		size_t j = fileMat.rfind(".mtx");
		if (j != std::string::npos)
			outputItem( fileMat.substr(0, j));
		else
			outputItem( fileMat);
	}

	// Dimension
	outputItem( A.m_n);
	//aNNZ 
	outputItem( A.m_nnz);

	cusparse::GPUTimer local_timer;
	local_timer.Start();
	hipsolverStatus_t status = A.QRSolve(x, y);
	local_timer.Stop();

	int code = status;

	// Error code
	if (status != HIPSOLVER_STATUS_SUCCESS) {
		outputItem(code, COLOR_RED);
		outputItem("");
		outputItem("");
		cout << "</tr>" << endl;
		return 1;
	} else
		outputItem(int(0));

	Abak.spmv(y, x_new);

	// The relative infinity norm of solution
	double nrm_target = cusp::blas::nrmmax(x);
	cusp::blas::axpy(x, x_new, (double)(-1));
	double rel_err = fabs(cusp::blas::nrmmax(x_new))/ nrm_target;
	if (isnan(cusp::blas::nrm1(x_new)))
		outputItem("NaN", COLOR_RED);
	else if (rel_err >= 1)
		outputItem(rel_err, COLOR_RED);
	else
		outputItem(rel_err);

	outputItem( local_timer.getElapsed());
	cout << "</tr>" << endl;
 

	return 0;
}
